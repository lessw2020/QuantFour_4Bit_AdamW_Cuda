#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
// All rights reserved.

// This is a productionized implementation of fused 4bit AdamW:
// "Memory Efficient Optimizers with 4-bit States"
// Bingrui Li, Jianfei Chen, Jun Zhu
// https://arxiv.org/abs/2309.01507

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/Exceptions.h>


using torch::Tensor;

static __device__ __const__ uint8_t _bitmask = 15;
static __device__ __const__ uint8_t _right_pack_bitmask = _bitmask << 4;

static __device__ __shared__ float _exp_reducer [64];

static __device__ __const__ float _exp_qmap [] = {
                -0.8875,
                -0.6625,
                -0.4375,
                -0.2125,
                -0.0775,
                -0.0325,
                -0.0055,
                0.0000,
                0.0055,
                0.0325,
                0.0775,
                0.2125,
                0.4375,
                0.6625,
                0.8875,
                1.0000,
};

static __device__ __const__ float _exp_qmidpt [] = {

            -0.775,
            -0.55,
            -0.325,
            -0.145,
            -0.055,
            -0.019,
            -0.00275,
            0.00275,
            0.019,
            0.055,
            0.145,
            0.325,
            0.55,
            0.775,
            0.94375,
};

static __device__ __const__ float _sq_qmap [] = {
                0.0625,
                0.1250,
                0.1875,
                0.2500,
                0.3125,
                0.3750,
                0.4375,
                0.5000,
                0.5625,
                0.6250,
                0.6875,
                0.7500,
                0.8125,
                0.8750,
                0.9375,
                1.0000,
};

static __device__ __const__ float _sq_qmidpt [] = {
            0.09375,
            0.15625,
            0.21875,
            0.28125,
            0.34375,
            0.40625,
            0.46875,
            0.53125,
            0.59375,
            0.65625,
            0.71875,
            0.78125,
            0.84375,
            0.90625,
            0.96875,
};

template <typename T>
__global__ void kernel_cuda_single_tensor(
        T* __restrict__ p,
        const T * __restrict__ g,
        T* __restrict__ exp_avg,
        T* __restrict__ exp_avg_sq,
        const float beta1,
        const float beta2,
        const float lr,
        const float weight_decay,
        const float eps,
        const float step,
        const size_t total_size)
{
        const int global_id = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_id >= total_size) return;

        float curr_grad = g[global_id];

        //decoupled weight decay
        p[global_id] = p[global_id] * (1 - lr * weight_decay);


        exp_avg[global_id] = beta1 * exp_avg[global_id] + (1 - beta1) * curr_grad;
        exp_avg_sq[global_id] = beta2 * exp_avg_sq[global_id] + (1 - beta2) * (curr_grad * curr_grad);

        const float correction1 = 1.0f - powf(beta1, step);
        const float correction2_sqrt = sqrtf(1.0f - powf(beta2, step));
        float step_size = lr / correction1;

        float denom = (sqrtf(exp_avg_sq[global_id]) / correction2_sqrt + eps); // * correction1;
        float update = (exp_avg[global_id]/denom); // + (weight_decay * p[global_id]);
        p[global_id] = p[global_id] - (step_size * update);
}

// interface and launcher for fused adamw cuda kernel
void cuda_fused_single_tensor(Tensor& p, Tensor& g, Tensor& exp_avg, Tensor& exp_avg_sq,
                      float beta1, float beta2, float lr, float weight_decay, float eps, float step) {
    // Get tensor size
    int total_size = p.numel();
    AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p),
              "parameter tensor is too large to be indexed with int32");

    const int block_dim = 128;
    int grid_dim = ((total_size + block_dim - 1) / block_dim);
    const dim3 blocks(grid_dim);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(p.scalar_type(), "cuda_fused_single_tensor", ([&] {
        kernel_cuda_single_tensor<scalar_t><<<blocks, block_dim>>>(
            p.data_ptr<scalar_t>(),
            g.data_ptr<scalar_t>(),
            exp_avg.data_ptr<scalar_t>(),
            exp_avg_sq.data_ptr<scalar_t>(),
            beta1,
            beta2,
            lr,
            weight_decay,
            eps,
            step,
            total_size
        );
    }));

    AT_CUDA_CHECK(hipGetLastError());
}

// binary search for quantization
__device__ __forceinline__ float q_mapping( const float* __restrict__ qmap,
                                            const float* __restrict__ qmidpt,
                                            float x)
{
    // 4 bit range
    int low = 0;
    int high = 15;

    if (x <= qmap[low]) return low;
    if (qmap[high] <=x) return high;

    #pragma unroll
    // replace with for loop?
    while (low < high) {
        int mid = (low + high) >> 1;
        if (qmap[mid] <= x)
        {
            low = mid + 1;
        }
        else
        {
            high = mid;
        }
    }

    return (qmidpt[low-1] < x) ? low : low-1;

}


// sequential threads parallel reduction to determine max value for each block for exp and sq
__device__ __forceinline__ void seq_threads_max_reducer(int tid, float* local_absmax_val) {

        _exp_reducer[tid]= *local_absmax_val;
        __syncthreads();

        // get to warp level memory
        if (tid < 32) {
            _exp_reducer[tid] = max(_exp_reducer[tid], _exp_reducer[tid + 32]);
        }
        __syncthreads();

        // then shuffle down warp synch
        if (tid < 16){
            #define Full_Mask 0xffffffff
            float val = _exp_reducer[tid];
            for (int offset = 16; offset > 0; offset /= 2)
                val = max(val, __shfl_down_sync(Full_Mask, val, offset));
            if (tid ==0) {
                *local_absmax_val = val;
            }
        }

}

/*
scratchpad
// warpReduction
        /*if (tid < 32) {
            _exp_reducer[tid] = max(_exp_reducer[tid], _exp_reducer[tid + 32]);
            _exp_reducer[tid] = max(_exp_reducer[tid], _exp_reducer[tid + 16]);
            _exp_reducer[tid] = max(_exp_reducer[tid], _exp_reducer[tid + 8]);
            _exp_reducer[tid] = max(_exp_reducer[tid], _exp_reducer[tid + 4]);
            _exp_reducer[tid] = max(_exp_reducer[tid], _exp_reducer[tid + 2]);
            _exp_reducer[tid] = max(_exp_reducer[tid], _exp_reducer[tid + 1]);

        }
        */


        // 0 32   0 16 0 8 0 4 0 2 0 1
        // 1 33   1 17 1 9 1 5 1 3 1 2
        // 31 63  31 47 31 39 31 35 31 33 31 32

        /*for (int s= 32; s > 16; s /=2) {
            if (tid < s) {
                _exp_reducer[tid] = max(_exp_reducer[tid], _exp_reducer[tid +s]);
            }
            //__syncthreads();
        }
        */



template <typename T>
__global__ void cuda_fused_4bit_kernel(
    T* __restrict__ p,
    const T* __restrict__ g,
    int8_t* __restrict__ exp,
    int8_t* __restrict__ sq,
    T* __restrict__ exp_qscale,
    T* __restrict__ sq_qscale,
    const float beta1,
    const float beta2,
    const float lr,
    const float weight_decay,
    const float eps,
    const float step,
    const size_t total_size,
    const float correction1,
    const float correction2_sqrt,
    const float step_size,
    const float weight_decay_update,
    const float resid_beta1,
    const float resid_beta2

)
{
    // establish spatial awareness
    const int thread_id = threadIdx.x;
    const int block_id = blockIdx.x;
    const int global_id = blockIdx.x * blockDim.x + thread_id;

    const int left_id = global_id << 1;
    const int right_id = left_id + 1;

    // fail fast
    if (left_id >= total_size) return;

    __shared__ float absmax_exp;
    __shared__ float absmax_sq;

    if (thread_id == 0) {
        absmax_exp = 0;
        absmax_sq = 0;
    }

    // left side processing -------------------------------------
    const int8_t exp_left_index = (exp[global_id]) & _bitmask;
    const int8_t sq_left_index = (sq[left_id]) & _bitmask;

    //decoupled weight decay
    p[left_id] = p[left_id] * weight_decay_update;

    // left exp and sq updates
    float curr_grad = g[left_id];
    float exp_avg_qscale = exp_qscale[block_id];

    T exp_left = _exp_qmap[exp_left_index] * exp_avg_qscale;
    exp_left = beta1 * exp_left + resid_beta1 * curr_grad;

    T sq_left = _sq_qmap[sq_left_index] * sq_qscale[block_id];
    sq_left = beta2 * sq_left + resid_beta2 * (curr_grad * curr_grad);

    //float denom = (sqrtf(sq_left) / correction2_sqrt + eps);
    //float update = (exp_left/denom);
    //float update = (exp_left/(sqrtf(sq_left) / correction2_sqrt + eps));

    // param update
    p[left_id] = p[left_id] - (step_size * (exp_left/(sqrtf(sq_left) / correction2_sqrt + eps)));

    // right side processing -------------------------------
    T exp_right =0;
    T sq_right = 0;

    if (right_id < total_size) {
        const int8_t exp_right_index = (exp[global_id] >> 4) & _bitmask;
        const int8_t sq_right_index = (sq[global_id]>>4) & _bitmask;
        curr_grad = g[right_id];

        //decoupled weight decay, right side
        p[right_id] = p[right_id] * weight_decay_update;

        exp_right = _exp_qmap[exp_right_index] * exp_avg_qscale;
        exp_right = beta1 * exp_right + resid_beta1 * curr_grad;

        sq_right = _sq_qmap[sq_right_index] * sq_qscale[block_id];
        sq_right = beta2 * sq_right + resid_beta2 * (curr_grad * curr_grad);

        //denom = (sqrtf(sq_right) / correction2_sqrt + eps);
        //update = (exp_right/denom);

        // param update
        p[right_id] = p[right_id] - (step_size * (exp_right/(sqrtf(sq_right) / correction2_sqrt + eps)));

        }

    // prepare quantization info - update absmax scales
    float local_absmax_exp = max((float)exp_left, (float)exp_right);
    float local_absmax_sq = max((float)sq_left, (float)sq_right);

    // --- sequential threads parallel reduction to
    // determine global absmax for exp
    seq_threads_max_reducer(thread_id, &local_absmax_exp);
    if (thread_id ==0) {
        exp_qscale[block_id] = local_absmax_exp; //_exp_reducer[0];
    }

    // same for sq
    seq_threads_max_reducer(thread_id, &local_absmax_sq);
    if (thread_id ==0) {
        sq_qscale[block_id] = local_absmax_sq; //_exp_reducer[0];
    }

    int8_t local_packed_exp = 0;
    int8_t local_packed_sq = 0;

    // quantize and pack
    const int8_t q_exp_left = (int8_t)q_mapping(_exp_qmap, _exp_qmidpt, (float)exp_left / absmax_exp);
    const int8_t q_sq_left = (int8_t)q_mapping(_sq_qmap, _sq_qmidpt, (float)sq_left / absmax_sq);
    local_packed_exp |= (q_exp_left & _bitmask);
    local_packed_sq |= (q_sq_left & _bitmask);

    if (right_id < total_size) {
        const int8_t q_exp_right = (int8_t)q_mapping(_exp_qmap, _exp_qmidpt, (float)exp_right / absmax_exp);
        const int8_t q_sq_right = (int8_t)q_mapping(_sq_qmap, _sq_qmidpt, (float)sq_right / absmax_sq);
        local_packed_exp |= (q_exp_right & _right_pack_bitmask);
        local_packed_sq |= (q_sq_right & _right_pack_bitmask);

    }

    // store updated exp and sq
    exp[global_id] = local_packed_exp;
    sq[global_id] = local_packed_sq;

    __syncthreads();

}

// interface and launcher for 4bit quantized cuda kernel
void cuda_fused_4bit(Tensor& p, Tensor& g,
                        Tensor& exp, Tensor& sq,
                        Tensor& exp_scale, Tensor& sq_scale,
                        float beta1, float beta2,
                        float lr, float weight_decay,
                        float eps, float step
                        ){

    int total_size = p.numel();
    const int block_size = 128;
    int grid = ((total_size + block_size -1) / block_size);
    const dim3 blocks(grid);
    //universal computations
    const float correction1 = 1.0f - powf(beta1, step);
    const float correction2_sqrt = sqrtf(1.0f - powf(beta2, step));
    const float step_size = lr / correction1;
    const float weight_decay_update = 1 - lr * weight_decay;
    const float resid_beta1 = 1.0f-beta1;
    const float resid_beta2 = 1.0f - beta2;


    AT_DISPATCH_FLOATING_TYPES_AND_HALF(p.scalar_type(), "cuda_fused_4bit", ([&] {
        cuda_fused_4bit_kernel<scalar_t><<<blocks, block_size/2>>>(
            p.data_ptr<scalar_t>(),
            g.data_ptr<scalar_t>(),
            exp.data_ptr<int8_t>(),
            sq.data_ptr<int8_t>(),
            exp_scale.data_ptr<scalar_t>(),
            sq_scale.data_ptr<scalar_t>(),
            beta1,
            beta2,
            lr,
            weight_decay,
            eps,
            step,
            total_size,
            correction1,
            correction2_sqrt,
            step_size,
            weight_decay_update,
            resid_beta1,
            resid_beta2

        );
    }));

    AT_CUDA_CHECK(hipGetLastError());
}
