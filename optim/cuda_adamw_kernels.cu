#include "hip/hip_runtime.h"

#include <ATen/ATen.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/Exceptions.h>

#include <torch/extension.h>
#include <THC/THCAtomics.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

using torch::Tensor;

void cuda_fused_single_tensor(Tensor& p, Tensor& g, Tensor& exp_avg, Tensor& exp_avg_sq,
                      float beta1, float beta2, float lr, float weight_decay, float eps, float step) {
    // Get tensor size
    int total_size = p.numel();
    AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p),
              "parameter tensor is too large to be indexed with int32");

    const int block_dim = 128;
    int grid_dim = ((total_size + block_dim - 1) / block_dim);
    const dim3 blocks(grid_dim);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(p.scalar_type(), "kernel_cuda_single_tensor", ([&] {
        adamw_cuda_kernel<scalar_t><<<blocks, block_dim>>>(
            p.data_ptr<scalar_t>(),
            g.data_ptr<scalar_t>(),
            exp_avg.data_ptr<scalar_t>(),
            exp_avg_sq.data_ptr<scalar_t>(),
            beta1,
            beta2,
            lr,
            weight_decay,
            eps,
            step,
            total_size
        );
    }));

    AT_CUDA_CHECK(hipGetLastError());
}


__device__ __forceinline__ float atomicMax(float * addr, float value) {

    return __int_as_float(atomicMax((int *)addr, __float_as_int(value)));
}

template <typename T>
__global__ void kernel_cuda_single_tensor(
        T* __restrict__ p,
        const T * __restrict__ g,
        T* __restrict__ exp_avg,
        T* __restrict__ exp_avg_sq,

        const float beta1,
        const float beta2,
        const float lr,
        const float weight_decay,
        const float eps,
        const float step,
        const size_t total_size)
{
        const int global_id = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_id >= total_size) return;

        exp_avg[global_id] = beta1 * exp_avg[global_id] + (1 - beta1) * g[global_id];
        exp_avg_sq[global_id] = beta2 * exp_avg_sq[global_id] + (1 - beta2) * g[global_id] * g[global_id];

        const float correction1 = 1.0f - powf(beta1, step);
        const float correction2_sqrt = sqrtf(1.0f - powf(beta2, step));

        float denom = (sqrtf(exp_avg_sq[global_id]) / correction2_sqrt + eps) * correction1;
        float update = (exp_avg[global_id]/denom) + (weight_decay * p[global_id]);
        p[global_id] = p[global_id] - (lr * update);
}
