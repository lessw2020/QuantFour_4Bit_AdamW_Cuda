#include "hip/hip_runtime.h"

#include <ATen/ATen.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/Exceptions.h>

#include <torch/extension.h>
#include <THC/THCAtomics.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

using torch::Tensor;

static __device__ __const__ float _exp_qmap [] = {
                -0.8875,
                -0.6625,
                -0.4375,
                -0.2125,
                -0.0775,
                -0.0325,
                -0.0055,
                0.0000,
                0.0055,
                0.0325,
                0.0775,
                0.2125,
                0.4375,
                0.6625,
                0.8875,
                1.0000,
};

static __device__ __const__ float _exp_qmidpt [] = {

            -0.775,
            -0.55,
            -0.325,
            -0.145,
            -0.055,
            -0.019,
            -0.00275,
            0.00275,
            0.019,
            0.055,
            0.145,
            0.325,
            0.55,
            0.775,
            0.94375,
};

static __device__ __const__ float _sq_qmap [] = {
                0.0625,
                0.1250,
                0.1875,
                0.2500,
                0.3125,
                0.3750,
                0.4375,
                0.5000,
                0.5625,
                0.6250,
                0.6875,
                0.7500,
                0.8125,
                0.8750,
                0.9375,
                1.0000,
};

static __device__ __const__ float _sq_qmidpt [] = {
            0.09375,
            0.15625,
            0.21875,
            0.28125,
            0.34375,
            0.40625,
            0.46875,
            0.53125,
            0.59375,
            0.65625,
            0.71875,
            0.78125,
            0.84375,
            0.90625,
            0.96875,
};

template <typename T>
__global__ void kernel_cuda_single_tensor(
        T* __restrict__ p,
        const T * __restrict__ g,
        T* __restrict__ exp_avg,
        T* __restrict__ exp_avg_sq,
        const float beta1,
        const float beta2,
        const float lr,
        const float weight_decay,
        const float eps,
        const float step,
        const size_t total_size)
{
        const int global_id = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_id >= total_size) return;

        float curr_grad = g[global_id];

        //decoupled weight decay
        p[global_id] = p[global_id] * (1 - lr * weight_decay);


        exp_avg[global_id] = beta1 * exp_avg[global_id] + (1 - beta1) * curr_grad;
        exp_avg_sq[global_id] = beta2 * exp_avg_sq[global_id] + (1 - beta2) * (curr_grad * curr_grad);

        const float correction1 = 1.0f - powf(beta1, step);
        const float correction2_sqrt = sqrtf(1.0f - powf(beta2, step));
        float step_size = lr / correction1;

        float denom = (sqrtf(exp_avg_sq[global_id]) / correction2_sqrt + eps); // * correction1;
        float update = (exp_avg[global_id]/denom); // + (weight_decay * p[global_id]);
        p[global_id] = p[global_id] - (step_size * update);
}

// interface and launcher for fused adamw cuda kernel
void cuda_fused_single_tensor(Tensor& p, Tensor& g, Tensor& exp_avg, Tensor& exp_avg_sq,
                      float beta1, float beta2, float lr, float weight_decay, float eps, float step) {
    // Get tensor size
    int total_size = p.numel();
    AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p),
              "parameter tensor is too large to be indexed with int32");

    const int block_dim = 128;
    int grid_dim = ((total_size + block_dim - 1) / block_dim);
    const dim3 blocks(grid_dim);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(p.scalar_type(), "cuda_fused_single_tensor", ([&] {
        kernel_cuda_single_tensor<scalar_t><<<blocks, block_dim>>>(
            p.data_ptr<scalar_t>(),
            g.data_ptr<scalar_t>(),
            exp_avg.data_ptr<scalar_t>(),
            exp_avg_sq.data_ptr<scalar_t>(),
            beta1,
            beta2,
            lr,
            weight_decay,
            eps,
            step,
            total_size
        );
    }));

    AT_CUDA_CHECK(hipGetLastError());
}

// binary search for quantization
__device__ __forceinline__ float q_mapping( const float* __restrict__ qmap,
                                            const float* __restrict__ qmidpt,
                                            float x)
{
    // 4 bit range
    int low = 0;
    int high = 15;

    if (x <= qmap[low]) return low;
    if (qmap[high] <=x) return high;

    while (low < high) {
        int mid = (low + high) >> 1;
        if (qmap[mid] <= x)
        {
            low = mid + 1;
        }
        else
        {
            high = mid;
        }
    }

    return (qmidpt[low-1] < x) ? low : low-1;

}

template <typename T>
__global__ void cuda_fused_4bit_kernel(
    T* __restrict__ p,
    const T* __restrict__ g,
    int8_t* __restrict__ exp,
    int8_t* __restrict__ sq,
    T* __restrict__ exp_qscale,
    T* __restrict__ sq_qscale,
    const float beta1,
    const float beta2,
    const float lr,
    const float weight_decay,
    const float eps,
    const float step,
    const size_t total_size,
    const float correction1,
    const float correction2_sqrt,
    const float step_size,
    const uint8_t bitmask

)
{
    // establish thread, block and global situational awareness
    const int thread_id = threadIdx.x;
    const int block_id = blockIdx.x;
    const int global_id = blockIdx.x * blockDim.x + thread_id;

    const int left_id = global_id << 1;
    const int right_id = left_id + 1;

    __shared__ float absmax_exp;
    __shared__ float absmax_sq;

    if (thread_id == 0) {
        absmax_exp = 0;
        absmax_sq = 0;
    }
    __syncthreads();

    if (left_id >= total_size) return;

    // left side processing
    const int8_t exp_left_index = (exp[global_id]) & bitmask;
    const int8_t sq_left_index = (sq[left_id]) & bitmask;

    //decoupled weight decay
    p[left_id] = p[left_id] * (1 - lr * weight_decay);

    // left exp and sq updates
    float curr_grad = g[left_id];
    float exp_avg_qscale = exp_qscale[block_id];

    T exp_left = _exp_qmap[exp_left_index] * exp_avg_qscale;
    exp_left = beta1 * exp_left + (1 - beta1) * curr_grad;

    T sq_left = _sq_qmap[sq_left_index] * sq_qscale[block_id];
    sq_left = beta2 * sq_left + (1 - beta2) * (curr_grad * curr_grad);

    float denom = (sqrtf(sq_left) / correction2_sqrt + eps);
    float update = (exp_left/denom);

    // param update
    p[left_id] = p[left_id] - (step_size * update);

    // right side processing
    T exp_right =0;
    T sq_right = 0;

    if (right_id < total_size) {
        const int8_t exp_right_index = (exp[global_id] >> 4) & bitmask;
        const int8_t sq_right_index = (sq[global_id]>>4) & bitmask;
        curr_grad = g[right_id];

        //decoupled weight decay, right side
        p[right_id] = p[right_id] * (1 - lr * weight_decay);

        exp_right = _exp_qmap[exp_right_index] * exp_avg_qscale;
        exp_right = beta1 * exp_right + (1-beta1) * curr_grad;

        sq_right = _sq_qmap[sq_right_index] * sq_qscale[block_id];
        sq_right = beta2 * sq_right + (1 - beta2) * (curr_grad * curr_grad);

        denom = (sqrtf(sq_right) / correction2_sqrt + eps);
        update = (exp_right/denom);

        // param update
        p[right_id] = p[right_id] - (step_size * update);

        }

    // prepare quantization info - update absmax scales
    float local_absmax_exp = fmax(fabsf((float)exp_left), fabsf((float)exp_right));
    float local_absmax_sq = fmaxf((float)sq_left, (float)sq_right);

    // determine global max for this block
    __int_as_float(atomicMax((int *)&absmax_exp, __float_as_int(local_absmax_exp)));
    __int_as_float(atomicMax((int *)&absmax_sq, __float_as_int(local_absmax_sq)));

    __syncthreads();

    int8_t local_packed_exp = 0;
    int8_t local_packed_sq = 0;

    // quantize and pack
    const int8_t q_exp_left = (int8_t)q_mapping(_exp_qmap, _exp_qmidpt, (float)exp_left / absmax_exp);
    const int8_t q_sq_left = (int8_t)q_mapping(_sq_qmap, _sq_qmidpt, (float)sq_left / absmax_sq);
    local_packed_exp |= (q_exp_left & bitmask);
    local_packed_sq |= (q_sq_left & bitmask);

    if (right_id < total_size) {
        const int8_t q_exp_right = (int8_t)q_mapping(_exp_qmap, _exp_qmidpt, (float)exp_right / absmax_exp);
        const int8_t q_sq_right = (int8_t)q_mapping(_sq_qmap, _sq_qmidpt, (float)sq_right / absmax_sq);
        local_packed_exp |= (q_exp_right & bitmask << 4);
        local_packed_sq |= (q_sq_right & bitmask << 4);

    }

    // store updated exp and sq
    exp[global_id] = local_packed_exp;
    sq[global_id] = local_packed_sq;

    if (thread_id == 0) {
        exp_qscale[block_id] = (T)absmax_exp;
        sq_qscale[block_id] = (T)absmax_sq;
    }
    __syncthreads();

}

// interface and launcher for 4bit quantized cuda kernel
void cuda_fused_4bit(Tensor& p, Tensor& g,
                        Tensor& exp, Tensor& sq,
                        Tensor& exp_scale, Tensor& sq_scale,
                        float beta1, float beta2,
                        float lr, float weight_decay,
                        float eps, float step
                        ){

    int total_size = p.numel();
    const int block_size = 128;
    int grid = ((total_size + block_size -1) / block_size);
    const dim3 blocks(grid);
    //universal computations
    const float correction1 = 1.0f - powf(beta1, step);
    const float correction2_sqrt = sqrtf(1.0f - powf(beta2, step));
    const float step_size = lr / correction1;

    // leverage constant memory
    const uint8_t g_bitmask = 15;


    AT_DISPATCH_FLOATING_TYPES_AND_HALF(p.scalar_type(), "cuda_fused_4bit", ([&] {
        cuda_fused_4bit_kernel<scalar_t><<<blocks, block_size/2>>>(
            p.data_ptr<scalar_t>(),
            g.data_ptr<scalar_t>(),
            exp.data_ptr<int8_t>(),
            sq.data_ptr<int8_t>(),
            exp_scale.data_ptr<scalar_t>(),
            sq_scale.data_ptr<scalar_t>(),
            beta1,
            beta2,
            lr,
            weight_decay,
            eps,
            step,
            total_size,
            correction1,
            correction2_sqrt,
            step_size,
            g_bitmask

        );
    }));

    AT_CUDA_CHECK(hipGetLastError());
}
